#include "hip/hip_runtime.h"
#define __MAKEMORE_COLONEL_CC__ 1

#include <stdio.h>
#include <assert.h>

#include "colonel.hh"

namespace makemore {

void enkv(const void *a, unsigned int n, void *da) {
  hipMemcpy(da, a, n, hipMemcpyHostToDevice);
}

void dekv(const void *da, unsigned int n, void *a) {
  hipMemcpy(a, da, n, hipMemcpyDeviceToHost);
}

void kmakev(void **dp, unsigned int n) {
  void *d = NULL;
  assert(n > 0);
  int ret = hipMalloc((void **)&d, n);
  assert(d != NULL);
  assert(ret == 0);
  *dp = d;
}

void kfreev(void *x) {
  hipFree(x);
}

void kzerov(void *x, unsigned int n) {
  hipMemset((void *)x, 0, n);
}

void kfill(double *x, unsigned int n, double v) {
  double *y = new double[n];
  for (unsigned int i = 0; i < n; ++i)
    y[i] = v;
  enk(y, n, x);
  delete[] y;
}

void kcopyv(const void *x, unsigned int n, void *y) {
  hipMemcpy(y, x, n, hipMemcpyDeviceToDevice);
}

__global__ void _kaddvec(const double *a, const double *b, unsigned int n, double *c) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

void kaddvec(const double *a, const double *b, unsigned int n, double *c) {
  int bs = 256;
  int gs = ((n + bs - 1) / bs);
  _kaddvec<<<gs, bs>>>(a, b, n, c);
}

__global__ void _ksubvec(const double *a, const double *b, unsigned int n, double *c) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] - b[i];
}

void ksubvec(const double *a, const double *b, unsigned int n, double *c) {
  int bs = 256;
  int gs = ((n + bs - 1) / bs);
  _ksubvec<<<gs, bs>>>(a, b, n, c);
}

__global__ void _ksumsq(
  const double *a, unsigned int n, double *sumsqp
) {
  unsigned int si = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int i0 = si * 128;
  if (i0 >= n)
    return;
  unsigned int i1 = (i0 + 128 >= n) ? n : (i0 + 128);
  
  double s = 0;
  for (unsigned int i = i0; i < i1; ++i)
    s += a[i] * a[i];
  sumsqp[si] = s;
}

double ksumsq(
  const double *a, unsigned int n
) {
  if (n == 0)
    return 0;

  double *sumsqp = NULL;
  unsigned int sumsqn = ((n + 127) / 128);
  kmake(&sumsqp, sumsqn);

  int bs = 128;
  int gs = (sumsqn + bs - 1) / bs;
  _ksumsq<<<gs, bs>>>(a, n, sumsqp);

  double *sumsqv = new double[sumsqn];
  dek(sumsqp, sumsqn, sumsqv);
  kfree(sumsqp);

  double s = 0;
  for (int i = 0; i < sumsqn; ++i)
    s += sumsqv[i];

  delete[] sumsqv;

  return s;
}

__global__ void _kmaxabs(
  const double *a, unsigned int n, double *maxp
) {
  unsigned int si = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int i0 = si * 128;
  if (i0 >= n)
    return;
  unsigned int i1 = (i0 + 128 >= n) ? n : (i0 + 128);

  unsigned int i = i0;
  double s = fabs(a[i]);
  ++i;
  
  for (; i < i1; ++i) {
    double aa = fabs(a[i]);
    if (aa > s)
      s = aa;
  }

  maxp[si] = s;
}

double kmaxabs(
  const double *a, unsigned int n
) {
  if (n == 0)
    return 0;

  double *maxp = NULL;
  unsigned int maxn = ((n + 127) / 128);
  kmake(&maxp, maxn);

  int bs = 128;
  int gs = (maxn + bs - 1) / bs;
  _kmaxabs<<<gs, bs>>>(a, n, maxp);

  double *maxv = new double[maxn];
  dek(maxp, maxn, maxv);

  double s = maxv[0];
  for (int i = 1; i < maxn; ++i)
    if (maxv[i] > s)
      s = maxv[i];

  kfree(maxp);
  delete[] maxv;

  return s;
}


__global__ void _kspliceadd(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n * xk)
    return;

  int ixk = i / n;
  int in = i % n;

  y[in * ym + ya + ixk] += x[in * xm + xa + ixk];
}

void kspliceadd(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  int bs = 256;
  int gs = ((n * xk + bs - 1) / bs);
  _kspliceadd<<<gs, bs>>>(x, n, xm, xa, xk, y, ym, ya);
}



__global__ void _ksplice(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n * xk)
    return;

  int ixk = i / n;
  int in = i % n;

  y[in * ym + ya + ixk] = x[in * xm + xa + ixk];
}

void ksplice(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  int bs = 256;
  int gs = ((n * xk + bs - 1) / bs);
  _ksplice<<<gs, bs>>>(x, n, xm, xa, xk, y, ym, ya);
}



__global__ void k_synth_conv(
  const double *in, int iw, int ih,
  double *out,
  int d, int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;


  int tmp = outi;
  int oz = tmp % oc; tmp /= oc;
  int ox = tmp % ow; tmp /= ow;
  int oy = tmp;

  int ix0 = ox - d;
  int ix1 = ox + d;
  int iy0 = oy - d;
  int iy1 = oy + d;

  int f = d * 2 + 1;
  // int wmvn = 3 * (oc + ic * f * f * oc);

  double outv = 0;

  for (int iyt = iy0; iyt <= iy1; ++iyt) {
    int iy = (iyt + ih) % ih;
    for (int ixt = ix0; ixt <= ix1; ++ixt) { 
      int ix = (ixt + iw) % iw;
      for (int iz = 0; iz < ic; ++iz) {
        int ini = iz + ic * (ix + iw * iy);

        int dx = ixt - ix0;
        int dy = iyt - iy0;
        int wi = oc + (iz + ic * (dx + f * (dy + f * oz)));

        double v = in[ini];
        double w = wmv[wi * 3];

        outv += v * w;
      }
    }
  }

  outv += wmv[oz * 3];

  out[outi] = outv;
}


int size_conv(
  int d, int ic, int oc
) {
  int f = d * 2 + 1;
  return (3 * (oc + ic * f * f * oc));
}



void synth_conv(
  const double *in, int iw, int ih,
  double *out,
  int d, int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_conv<<<gs, bs>>>(
    in, iw, ih, out, d, ic, oc, wmv
  );
}

__global__ void k_learn_conv1(
  const double *in, int iw, int ih,
  const double *fout,

  int d, int ic, int oc,

  double *wmv,
  double nu, double b1, double b2
) {
  if (!(nu > 0))
    return;

  int f = d * 2 + 1;
  int wn = (oc + ic * f * f * oc);

  int wi = blockIdx.x * blockDim.x + threadIdx.x;
  if (wi >= wn)
    return;

  int ow = iw;
  int oh = ih;

  double dw = 0;

  if (wi < oc) {
    int oz = wi;

    for (int oy = 0; oy < oh; ++oy) {
      for (int ox = 0; ox < ow; ++ox) {
        int outi = oz + oc * (ox + ow * oy);
        dw += fout[outi];
      }
    }
  } else {
    int tmp = wi - oc;
    int iz = tmp % ic; tmp /= ic;
    int dx = tmp % f; tmp /= f;
    int dy = tmp % f; tmp /= f;
    int oz = tmp % oc;

    for (int oy = 0; oy < oh; ++oy) {
      for (int ox = 0; ox < ow; ++ox) {
        int ix = (ox - d + dx + iw) % iw;
        int iy = (oy - d + dy + ih) % ih;

        int ini = iz + ic * (ix + iw * iy);
        int outi = oz + oc * (ox + ow * oy);

        dw += fout[outi] * in[ini];
      }
    }
  }

  double m = wmv[wi * 3 + 1];
  double v = wmv[wi * 3 + 2];

  m = (1.0 - b1) * m + b1 * dw;
  v = (1.0 - b2) * v + b2 * dw * dw;

  wmv[wi * 3 + 1] = m;
  wmv[wi * 3 + 2] = v;
}


__global__ void k_learn_conv2(
  double *fin, int iw, int ih,
  const double *fout,

  int d, int ic, int oc,

  const double *wmv
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int ow = iw;
  int oh = ih;

  int tmp = ini;
  int iz = tmp % ic; tmp /= ic;
  int ix = tmp % iw; tmp /= iw;
  int iy = tmp;

  int ox0 = ix - d;
  int ox1 = ix + d;
  int oy0 = iy - d;
  int oy1 = iy + d;

  int f = d * 2 + 1;
  // int wmvn = 3 * (oc + ic * f * f * oc);

  double finv = 0;

  for (int oyt = oy0; oyt <= oy1; ++oyt) {
    int oy = (oyt + oh) % oh;
    for (int oxt = ox0; oxt <= ox1; ++oxt) { 
      int ox = (oxt + ow) % ow;
      for (int oz = 0; oz < oc; ++oz) {
        int outi = oz + oc * (ox + ow * oy);

        int dx = ix - oxt + d;
        int dy = iy - oyt + d;
        int wi = oc + (iz + ic * (dx + f * (dy + f * oz)));

        double v = fout[outi];
        double w = wmv[wi * 3];
        finv += v * w;
      }
    }
  }

  fin[ini] = finv;
}


__global__ void k_learn_conv3(
  int d, int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  if (!(nu > 0))
    return;

  int f = d * 2 + 1;
  int wn = (oc + ic * f * f * oc);

  int wi = blockIdx.x * blockDim.x + threadIdx.x;
  if (wi >= wn)
    return;

  double w = wmv[wi * 3 + 0];
  double m = wmv[wi * 3 + 1];
  double v = wmv[wi * 3 + 2];

  if (rounds < 1024.0) {
    m = m / (1.0 - pow(1.0 - b1, 1.0 + rounds));
    v = v / (1.0 - pow(1.0 - b2, 1.0 + rounds));
  }

  w += nu * m / (sqrt(v) + eps);

  wmv[wi * 3 + 0] = w;
}

void learn_conv(
  double *fin, int iw, int ih,
  const double *fout,

  int d, int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  int f = d * 2 + 1;
  int wn = (oc + ic * f * f * oc);
  int inn = iw * ih * ic;

  int bs = 256;
  int gsw = (wn + bs - 1) / bs;
  int gsi = (inn + bs - 1) / bs;

  k_learn_conv1<<<gsw, bs>>>(fin, iw, ih, fout, d, ic, oc, wmv, nu, b1, b2);
  k_learn_conv2<<<gsi, bs>>>(fin, iw, ih, fout, d, ic, oc, wmv);
  k_learn_conv3<<<gsw, bs>>>(d, ic, oc, wmv, nu, b1, b2, eps, rounds);
}


__global__ void k_synth_pad(
  const double *in, int iw, int ih,
  double *out,
  int ic, int oc, const double *kbuf
) {
  int ow = iw;
  int oh = ih;

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;
  int tmp = outi;
  int oz = tmp % oc; tmp /= oc;
  int ox = tmp % ow; tmp /= ow;
  int oy = tmp;

  int ix = ox;
  int iy = oy;

  if (oz < ic) {
    int iz = oz;
    int ini = iz + ic * (ix + iw * iy);
    out[outi] = in[ini];
  } else {
    int kbufi = (oz - ic) + (oc - ic) * (ox + ow * oy);
    out[outi] = kbuf[kbufi];
  }

}

void synth_pad(
  const double *in, int iw, int ih,
  double *out,
  int ic, int oc, const double *kbuf
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_pad<<<gs, bs>>>(
    in, iw, ih, out, ic, oc, kbuf
  );
}

__global__ void k_learn_pad(
  double *fin, int iw, int ih,
  const double *fout,
  int ic, int oc
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int ow = iw;

  int tmp = ini;
  int iz = tmp % ic; tmp /= ic;
  int ix = tmp % iw; tmp /= iw;
  int iy = tmp;

  int oz = iz;
  int ox = ix;
  int oy = iy;

  int outi = oz + oc * (ox + ow * oy);
  fin[ini] = fout[outi];
}

void learn_pad(
  double *fin, int iw, int ih,
  const double *fout,

  int ic, int oc
) {
  int inn = iw * ih * ic;

  int bs = 256;
  int gs = (inn + bs - 1) / bs;
  k_learn_pad<<<gs, bs>>>(
    fin, iw, ih, fout, ic, oc
  );
}

__global__ void k_synth_relu(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  int ini = outi;
  if (in[ini] > 0)
    out[outi] = in[ini];
  else
    out[outi] = 0;
}

void synth_relu(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_relu<<<gs, bs>>>(
    in, iw, ih, out, ic
  );
}

__global__ void k_learn_relu(
  double *fin, int iw, int ih,
  const double *fout,
  int ic
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int outi = ini;

  if (fin[ini] > 0) {
    fin[ini] = fout[outi];
  } else {
    fin[ini] = 0;
  }
}

void learn_relu(
  double *fin, int iw, int ih,
  const double *fout,

  int ic
) {
  int inn = iw * ih * ic;

  int bs = 256;
  int gs = (inn + bs - 1) / bs;
  k_learn_relu<<<gs, bs>>>(
    fin, iw, ih, fout, ic
  );
}


__global__ void k_synth_abs(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  int ini = outi;
  if (in[ini] > 0)
    out[outi] = in[ini];
  else
    out[outi] = -in[ini];
}

void synth_abs(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_abs<<<gs, bs>>>(
    in, iw, ih, out, ic
  );
}

__global__ void k_learn_abs(
  double *fin, int iw, int ih,
  const double *fout,
  int ic
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int outi = ini;

  if (fin[ini] > 0) {
    fin[ini] = fout[outi];
  } else {
    fin[ini] = -fout[outi];
  }
}

void learn_abs(
  double *fin, int iw, int ih,
  const double *fout,

  int ic
) {
  int inn = iw * ih * ic;

  int bs = 256;
  int gs = (inn + bs - 1) / bs;
  k_learn_abs<<<gs, bs>>>(
    fin, iw, ih, fout, ic
  );
}


__global__ void k_synth_upscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  int ow = (iw << s);
  int oh = (ih << s);

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  int tmp = outi;
  int oz = tmp % oc; tmp /= oc;
  int ox = tmp % ow; tmp /= ow;
  int oy = tmp;

  // assert((oc << (s + s)) == ic);
  int ix = (ox >> s);
  int iy = (oy >> s);

  int f = (1 << s);
  int dx = ox % f;
  int dy = oy % f;

  int iz = oz + oc * (dx + f * dy);
  int ini = iz + ic * (ix + iw * iy);

  out[outi] = in[ini];
}

void synth_upscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  assert((oc << (s + s)) == ic);

  int ow = (iw << s);
  int oh = (ih << s);
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_upscale<<<gs, bs>>>(
    in, iw, ih, out, s, ic, oc
  );
}

__global__ void k_learn_upscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int ow = (iw << s);
  // int oh = (ih << s);

  int tmp = ini;
  int iz = tmp % ic; tmp /= ic;
  int ix = tmp % iw; tmp /= iw;
  int iy = tmp;

  int f = (1 << s);
  tmp = iz;
  int oz = tmp % oc; tmp /= oc;
  int dx = tmp % f; tmp /= f;
  int dy = tmp;
  int ox = (ix << s) + dx;
  int oy = (iy << s) + dy;

  int outi = oz + oc * (ox + ow * oy);

  fin[ini] = fout[outi];
}

void learn_upscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;

  int bs = 256;
  int gs = (inn + bs - 1) / bs;

  k_learn_upscale<<<gs, bs>>>(fin, iw, ih, fout, s, ic, oc);
}



__global__ void k_synth_downscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  int ow = (iw >> s);
  int oh = (ih >> s);

  int outn = ow * oh * oc;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  int tmp = outi;
  int oz = tmp % oc; tmp /= oc;
  int ox = tmp % ow; tmp /= ow;
  int oy = tmp;

  int f = (1 << s);
  tmp = oz;
  int iz = tmp % ic; tmp /= ic;
  int dx = tmp % f; tmp /= f;
  int dy = tmp;

  int ix = (ox << s) + dx;
  int iy = (oy << s) + dy;

  int ini = iz + ic * (ix + iw * iy);

  out[outi] = in[ini];
}

void synth_downscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  int bs = 256;
  int gs = (outn + bs - 1) / bs;
  k_synth_downscale<<<gs, bs>>>(
    in, iw, ih, out, s, ic, oc
  );
}


__global__ void k_learn_downscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;
  int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  int ow = (iw >> s);
  // int oh = (ih >> s);

  int tmp = ini;
  int iz = tmp % ic; tmp /= ic;
  int ix = tmp % iw; tmp /= iw;
  int iy = tmp;

  int f = (1 << s);
  int ox = (ix >> s);
  int oy = (iy >> s);

  int dx = ix % f;
  int dy = iy % f;
  int oz = iz + ic * (dx + f * dy);

  int outi = oz + oc * (ox + ow * oy);

  fin[ini] = fout[outi];
}

void learn_downscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;

  int bs = 256;
  int gs = (inn + bs - 1) / bs;

  k_learn_downscale<<<gs, bs>>>(fin, iw, ih, fout, s, ic, oc);
}





}
