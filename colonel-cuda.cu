#include "hip/hip_runtime.h"
#define __MAKEMORE_COLONEL_CU__ 1

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <math.h>

#include "colonel.hh"

namespace makemore {

int kdev = 0;
int kbs = 256;

void setkdev(int i) {
  assert(i >= 0);
  assert(i <= kndevs());
  if (i > 0)
    assert(0 == hipSetDevice(i - 1));
  kdev = i;
}

void setkbs(int i) {
  assert(i > 0);
  kbs = i;
}

int kndevs() {
  int ndevs = 0;
  hipGetDeviceCount(&ndevs);
  assert(ndevs >= 0);
  return (1 + ndevs);
}


#undef syncthreads
#define syncthreads() __syncthreads()

#undef DEFN_KERNEL
#define DEFN_KERNEL(f, args...) \
  __global__ void _gpu_ ## f(long __n, args)
#undef PREF_KERNEL
#define PREF_KERNEL \
  long i = blockIdx.x * blockDim.x + threadIdx.x; \
  if (i >= __n) \
    return;

#undef PRE
#define PRE(x) _gpu_ ## x

#include "colonel-core.inc"



#undef syncthreads
#define syncthreads() assert(!"no syncthreads in cpu mode")

#undef __device__
#define __device__ 

#undef PRE
#define PRE(x) _cpu_ ## x


#undef DEFN_KERNEL
#define DEFN_KERNEL(f, args...) \
  void _cpu_ ## f (long i, long __n, args)
#undef PREF_KERNEL
#define PREF_KERNEL \
  if (i >= __n) \
    return;

#include "colonel-core.inc"



#define CALL_KERNEL(f, _n, args...) do { \
  long __n = (_n); \
  if (kdev) { \
    int __bs = kbs, __gs = ((__n + __bs - 1) / __bs); \
    _gpu_ ## f <<<__gs, __bs>>>(__n, args); \
  } else { \
    for (long __i = __n - 1; __i >= 0; --__i) { \
      _cpu_ ## f (__i, __n, args); \
    } \
  } \
} while (0);



void enkv(const void *a, unsigned int n, void *da) {
  if (kdev) {
    ::hipMemcpy(da, a, n, hipMemcpyHostToDevice);
  } else {
    ::memcpy(da, a, n);
  }
}

void dekv(const void *da, unsigned int n, void *a) {
  if (kdev)
    ::hipMemcpy(a, da, n, hipMemcpyDeviceToHost);
  else
    ::memcpy(a, da, n);
}

void kmakev(void **dp, unsigned int n) {
  if (kdev) {
    void *d = NULL;
    // assert(n > 0);
    int ret = ::hipMalloc((void **)&d, n);
    // assert(d != NULL);
    assert(ret == 0);
    *dp = d;
  } else {
    *dp = (void *)(new char[n]);
    assert(*dp);
  }
}

void kfreev(void *x) {
  if (kdev)
    ::hipFree(x);
  else
    delete[] ((char *)x);
}

void kzerov(void *x, unsigned int n) {
  if (kdev)
    ::hipMemset((void *)x, 0, n);
  else
    ::memset(x, 0, n);
}

void kfill(double *x, unsigned int n, double v) {
  if (kdev) {
    double *y = new double[n];
    for (unsigned int i = 0; i < n; ++i)
      y[i] = v;
    enk(y, n, x);
    delete[] y;
  } else {
    for (unsigned int i = 0; i < n; ++i)
      x[i] = v;
  }
}

void kcopyv(const void *x, unsigned int n, void *y) {
  if (kdev)
    ::hipMemcpy(y, x, n, hipMemcpyDeviceToDevice);
  else
    ::memcpy(y, x, n);
}

#include "colonel-common.inc"

}
