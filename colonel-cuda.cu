#include "hip/hip_runtime.h"
#define __MAKEMORE_COLONEL_CU__ 1

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <math.h>

#include "colonel.hh"

namespace makemore {

int kdev = 0;
int kbs = 256;

void setkdev(int i) {
  assert(i >= 0);
  assert(i <= kndevs());
  if (i > 0)
    assert(0 == hipSetDevice(i - 1));
  kdev = i;
}

void setkbs(int i) {
  assert(i > 0);
  kbs = i;
}

int kndevs() {
  int ndevs = 0;
  hipGetDeviceCount(&ndevs);
  assert(ndevs >= 0);
  return (1 + ndevs);
}



#undef DEFN_KERNEL
#define DEFN_KERNEL(f, args...) \
  __global__ void _gpu_ ## f(long __n, args)
#undef PREF_KERNEL
#define PREF_KERNEL \
  long i = blockIdx.x * blockDim.x + threadIdx.x; \
  if (i >= __n) \
    return;

#include "colonel.inc"



#undef DEFN_KERNEL
#define DEFN_KERNEL(f, args...) \
  void _cpu_ ## f (long i, long __n, args)
#undef PREF_KERNEL
#define PREF_KERNEL \
  if (i >= __n) \
    return;

#include "colonel.inc"



#define CALL_KERNEL(f, _n, args...) do { \
  long __n = (_n); \
  if (kdev) { \
    int __bs = kbs, __gs = ((__n + __bs - 1) / __bs); \
    _gpu_ ## f <<<__gs, __bs>>>(__n, args); \
  } else { \
    for (long __i = __n - 1; __i >= 0; --__i) { \
      _cpu_ ## f (__i, __n, args); \
    } \
  } \
} while (0);



void enkv(const void *a, unsigned int n, void *da) {
  if (kdev) {
    ::hipMemcpy(da, a, n, hipMemcpyHostToDevice);
  } else {
    ::memcpy(da, a, n);
  }
}

void dekv(const void *da, unsigned int n, void *a) {
  if (kdev)
    ::hipMemcpy(a, da, n, hipMemcpyDeviceToHost);
  else
    ::memcpy(a, da, n);
}

void kmakev(void **dp, unsigned int n) {
  if (kdev) {
    void *d = NULL;
    // assert(n > 0);
    int ret = ::hipMalloc((void **)&d, n);
    // assert(d != NULL);
    assert(ret == 0);
    *dp = d;
  } else {
    *dp = (void *)(new char[n]);
    assert(*dp);
  }
}

void kfreev(void *x) {
  if (kdev)
    ::hipFree(x);
  else
    delete[] ((char *)x);
}

void kzerov(void *x, unsigned int n) {
  if (kdev)
    ::hipMemset((void *)x, 0, n);
  else
    ::memset(x, 0, n);
}

void kfill(double *x, unsigned int n, double v) {
  if (kdev) {
    double *y = new double[n];
    for (unsigned int i = 0; i < n; ++i)
      y[i] = v;
    enk(y, n, x);
    delete[] y;
  } else {
    for (unsigned int i = 0; i < n; ++i)
      x[i] = v;
  }
}

void kcopyv(const void *x, unsigned int n, void *y) {
  if (kdev)
    ::hipMemcpy(y, x, n, hipMemcpyDeviceToDevice);
  else
    ::memcpy(y, x, n);
}

void kaddvec(const double *a, const double *b, unsigned int n, double *c) {
  CALL_KERNEL(kaddvec, n, a, b, n, c);
}

void ksubvec(const double *a, const double *b, unsigned int n, double *c) {
  CALL_KERNEL(ksubvec, n, a, b, n, c);
}

double ksumsq(
  const double *a, unsigned int n
) {
  if (n == 0)
    return 0;

  double *sumsqp = NULL;
  unsigned int sumsqn = ((n + 127) / 128);
  kmake(&sumsqp, sumsqn);

  CALL_KERNEL(ksumsq, sumsqn, a, n, sumsqp);

  double *sumsqv = new double[sumsqn];
  dek(sumsqp, sumsqn, sumsqv);
  kfree(sumsqp);

  double s = 0;
  for (int i = 0; i < sumsqn; ++i)
    s += sumsqv[i];

  delete[] sumsqv;

  return s;
}

double kmaxabs(
  const double *a, unsigned int n
) {
  if (n == 0)
    return 0;

  double *maxp = NULL;
  unsigned int maxn = ((n + 127) / 128);
  kmake(&maxp, maxn);

  CALL_KERNEL(kmaxabs, maxn, a, n, maxp);

  double *maxv = new double[maxn];
  dek(maxp, maxn, maxv);

  double s = maxv[0];
  for (int i = 1; i < maxn; ++i)
    if (maxv[i] > s)
      s = maxv[i];

  kfree(maxp);
  delete[] maxv;

  return s;
}


void kspliceadd(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  CALL_KERNEL(kspliceadd, n * xk, x, n, xm, xa, xk, y, ym, ya);
}



void ksplice(
  const double *x, int n, int xm, int xa, int xk,
  double *y, int ym, int ya
) {
  CALL_KERNEL(ksplice, n * xk, x, n, xm, xa, xk, y, ym, ya);
}

int size_norm(
  int ic, int ow, int oh, int oc
) {
  return (2 * 3 * (ow * oh * oc));
}

void synth_norm(
  const double *in, int iw, int ih,
  double *out,
  int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_norm, outn,
    in, iw, ih, out, ic, oc, wmv
  );
}


void learn_norm(
  double *fin, int iw, int ih,
  const double *fout,

  int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  int outn = iw * ih * oc;
  int wn = outn * 2;
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_norm1, wn, fin, iw, ih, fout, ic, oc, wmv, nu, b1, b2);
  CALL_KERNEL(learn_norm2, inn, fin, iw, ih, fout, ic, oc, wmv);
  CALL_KERNEL(learn_norm3, wn, iw, ih, ic, oc, wmv, nu, b1, b2, eps, rounds);
}

int size_bias(
  int ic, int ow, int oh, int oc
) {
  return (3 * (ow * oh * oc));
}

void synth_bias(
  const double *in, int iw, int ih,
  double *out,
  int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_bias, outn,
    in, iw, ih, out, ic, oc, wmv
  );
}


void learn_bias(
  double *fin, int iw, int ih,
  const double *fout,

  int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  int outn = iw * ih * oc;
  int wn = outn;
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_bias1, wn, fin, iw, ih, fout, ic, oc, wmv, nu, b1, b2);
  CALL_KERNEL(learn_bias2, inn, fin, iw, ih, fout, ic, oc, wmv);
  CALL_KERNEL(learn_bias3, wn, iw, ih, ic, oc, wmv, nu, b1, b2, eps, rounds);
}

int size_local(
  int d, int ic, int ow, int oh, int oc
) {
  int f = d * 2 + 1;
  return (3 * (ow * oh * oc * (1 + ic * f * f)));
}

void synth_local(
  const double *in, int iw, int ih,
  double *out,
  int d, int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_local, outn,
    in, iw, ih, out, d, ic, oc, wmv
  );
}


void learn_local(
  double *fin, int iw, int ih,
  const double *fout,

  int d, int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  int f = d * 2 + 1;
  int outn = iw * ih * oc;
  int wn = (outn + ic * f * f * outn);
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_local1, wn, fin, iw, ih, fout, d, ic, oc, wmv, nu, b1, b2);
  CALL_KERNEL(learn_local2, inn, fin, iw, ih, fout, d, ic, oc, wmv);
  CALL_KERNEL(learn_local3, wn, iw, ih, d, ic, oc, wmv, nu, b1, b2, eps, rounds);
}


int size_conv(
  int d, int ic, int oc
) {
  int f = d * 2 + 1;
  return (3 * (oc + ic * f * f * oc));
}



void synth_conv(
  const double *in, int iw, int ih,
  double *out,
  int d, int ic, int oc,
  const double *wmv
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_conv, outn,
    in, iw, ih, out, d, ic, oc, wmv
  );
}


void learn_conv(
  double *fin, int iw, int ih,
  const double *fout,

  int d, int ic, int oc,

  double *wmv,
  double nu, double b1, double b2, double eps, double rounds
) {
  int f = d * 2 + 1;
  int wn = (oc + ic * f * f * oc);
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_conv1, wn, fin, iw, ih, fout, d, ic, oc, wmv, nu, b1, b2);
  CALL_KERNEL(learn_conv2, inn, fin, iw, ih, fout, d, ic, oc, wmv);
  CALL_KERNEL(learn_conv3, wn, d, ic, oc, wmv, nu, b1, b2, eps, rounds);
}


void synth_pad(
  const double *in, int iw, int ih,
  double *out,
  int ic, int oc, const double *kbuf
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_pad, outn,
    in, iw, ih, out, ic, oc, kbuf
  );
}


void learn_pad(
  double *fin, int iw, int ih,
  const double *fout,

  int ic, int oc
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_pad, inn,
    fin, iw, ih, fout, ic, oc
  );
}

void synth_sigm(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_sigm, outn,
    in, iw, ih, out, ic
  );
}

void learn_sigm(
  double *fin, int iw, int ih,
  const double *fout,

  int ic
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_sigm, inn,
    fin, iw, ih, fout, ic
  );
}

void synth_relu(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_relu, outn,
    in, iw, ih, out, ic
  );
}

void learn_relu(
  double *fin, int iw, int ih,
  const double *fout,

  int ic
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_relu, inn,
    fin, iw, ih, fout, ic
  );
}


void synth_abs(
  const double *in, int iw, int ih,
  double *out,
  int ic
) {
  int ow = iw;
  int oh = ih;
  int oc = ic;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_abs, outn,
    in, iw, ih, out, ic
  );
}

void learn_abs(
  double *fin, int iw, int ih,
  const double *fout,

  int ic
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_abs, inn,
    fin, iw, ih, fout, ic
  );
}


void synth_upscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  assert((oc << (s + s)) == ic);

  int ow = (iw << s);
  int oh = (ih << s);
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_upscale, outn,
    in, iw, ih, out, s, ic, oc
  );
}


void learn_upscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_upscale, inn, fin, iw, ih, fout, s, ic, oc);
}




void synth_downscale(
  const double *in, int iw, int ih,
  double *out,
  int s, int ic, int oc
) {
  int ow = iw;
  int oh = ih;
  int outn = ow * oh * oc;

  CALL_KERNEL(synth_downscale, outn,
    in, iw, ih, out, s, ic, oc
  );
}


void learn_downscale(
  double *fin, int iw, int ih,
  const double *fout,
  int s, int ic, int oc
) {
  int inn = iw * ih * ic;

  CALL_KERNEL(learn_downscale, inn, fin, iw, ih, fout, s, ic, oc);
}


}
